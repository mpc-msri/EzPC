#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"
#include <cassert>
#include "gpu_ctx.h"

extern "C" GPUContext* initialize_gpu_ctx() {
    GPUContext* c = new GPUContext;
    checkCudaErrors(hipSetDevice(0));
    checkCudaErrors(hipStreamCreateWithFlags(&(c->stream), hipStreamNonBlocking));
    int isMemPoolSupported = 0;

    checkCudaErrors(hipDeviceGetAttribute(&isMemPoolSupported,
                                         hipDeviceAttributeMemoryPoolsSupported, 0));
    // printf("%d\n", isMemPoolSupported);
    assert(isMemPoolSupported);
    /* implicitly assumes that the device is 0 */    
    checkCudaErrors(hipDeviceGetDefaultMemPool(&(c->mempool), 0));
    uint64_t threshold = UINT64_MAX;
    checkCudaErrors(hipMemPoolSetAttribute(c->mempool, hipMemPoolAttrReleaseThreshold, &threshold));
    uint64_t* d_dummy_ptr;
    uint64_t bytes = 15 * (1ULL << 30);
    checkCudaErrors(hipMallocAsync(&d_dummy_ptr, bytes, c->stream));
    checkCudaErrors(hipFreeAsync(d_dummy_ptr, c->stream));    
    uint64_t reserved_read, threshold_read;
    checkCudaErrors(hipMemPoolGetAttribute(c->mempool, hipMemPoolAttrReservedMemCurrent, &reserved_read));
    checkCudaErrors(hipMemPoolGetAttribute(c->mempool, hipMemPoolAttrReleaseThreshold, &threshold_read));
    printf("reserved memory: %lu %lu\n", reserved_read, threshold_read);
    return c;
}
