#include "hip/hip_runtime.h"

/*
 * Copyright (c) Meta Platforms, Inc. and its affiliates. All Rights Reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include <cstdint>
#include <cstdio>
#include "hip/hip_runtime_api.h"
// #include "gpu_data_types.h"

#define block_size 128

// namespace aes {

// This AES implementation is based on
// https://github.com/kokke/tiny-AES-c/blob/master/aes.c
// authored by kokke and et al. and distributed under public domain license.
//
// This is free and unencumbered software released into the public domain.
//
// Anyone is free to copy, modify, publish, use, compile, sell, or
// distribute this software, either in source code form or as a compiled
// binary, for any purpose, commercial or non-commercial, and by any
// means.
//
// In jurisdictions that recognize copyright laws, the author or authors
// of this software dedicate any and all copyright interest in the
// software to the public domain. We make this dedication for the benefit
// of the public at large and to the detriment of our heirs and
// successors. We intend this dedication to be an overt act of
// relinquishment in perpetuity of all present and future rights to this
// software under copyright law.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
// MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
// IN NO EVENT SHALL THE AUTHORS BE LIABLE FOR ANY CLAIM, DAMAGES OR
// OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
// ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
// OTHER DEALINGS IN THE SOFTWARE.
//
// For more information, please refer to <http://unlicense.org/>
//
// Adapted for CUDA by Pavel Belevich

/*****************************************************************************/
/* Defines:                                                                  */
/*****************************************************************************/
#define TORCH_CSPRNG_HOST_DEVICE __host__ __device__
#define TORCH_CSPRNG_CONSTANT __constant__

// The number of columns comprising a state in AES. This is a constant in AES. Value=4
#define Nb 4

#if defined(AES256) && (AES256 == 1)
#define Nk 8
#define Nr 14
#elif defined(AES192) && (AES192 == 1)
#define Nk 6
#define Nr 12
#else
#define Nk 4  // The number of 32 bit words in a key.
#define Nr 10 // Neha: Changed this from 10 to 11       // The number of rounds in AES Cipher.
#endif

constexpr size_t block_t_size = 16;

typedef uint8_t state_t[4][4];

// The lookup-tables are marked const so they can be placed in read-only storage instead of RAM
// The numbers below can be computed dynamically trading ROM for RAM -
// This can be useful in (embedded) bootloader applications, where ROM is often limited.
TORCH_CSPRNG_CONSTANT const uint8_t sbox[256] = {
    //0     1    2      3     4    5     6     7      8    9     A      B    C     D     E     F
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16};

TORCH_CSPRNG_CONSTANT const uint8_t rsbox[256] = {
    0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
    0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
    0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
    0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
    0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
    0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
    0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
    0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
    0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
    0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
    0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
    0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
    0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
    0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
    0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
    0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d};

// The round constant word array, Rcon[i], contains the values given by
// x to the power (i-1) being powers of x (x is denoted as {02}) in the field GF(2^8)
TORCH_CSPRNG_CONSTANT const uint8_t Rcon[11] = {
    0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36};

TORCH_CSPRNG_CONSTANT /*const*/ uint8_t d_RoundKeys[4 * 176];
uint8_t h_RoundKeys[4][176];

#define getSBoxValue(num) (sbox[(num)])

#define getSBoxInvert(num) (rsbox[(num)])

// This function produces Nb(Nr+1) round keys. The round keys are used in each round to decrypt the states.
TORCH_CSPRNG_HOST_DEVICE void KeyExpansion(uint8_t *RoundKey, const uint8_t *Key)
{
  unsigned int i, j, k;
  uint8_t tempa[4]; // Used for the column/row operations

  // The first round key is the key itself.
  for (i = 0; i < Nk; ++i)
  {
    RoundKey[(i * 4) + 0] = Key[(i * 4) + 0];
    RoundKey[(i * 4) + 1] = Key[(i * 4) + 1];
    RoundKey[(i * 4) + 2] = Key[(i * 4) + 2];
    RoundKey[(i * 4) + 3] = Key[(i * 4) + 3];
  }

  // All other round keys are found from the previous round keys.
  for (i = Nk; i < Nb * (Nr + 1); ++i)
  {
    {
      k = (i - 1) * 4;
      tempa[0] = RoundKey[k + 0];
      tempa[1] = RoundKey[k + 1];
      tempa[2] = RoundKey[k + 2];
      tempa[3] = RoundKey[k + 3];
    }

    if (i % Nk == 0)
    {
      // This function shifts the 4 bytes in a word to the left once.
      // [a0,a1,a2,a3] becomes [a1,a2,a3,a0]

      // Function RotWord()
      {
        const uint8_t u8tmp = tempa[0];
        tempa[0] = tempa[1];
        tempa[1] = tempa[2];
        tempa[2] = tempa[3];
        tempa[3] = u8tmp;
      }

      // SubWord() is a function that takes a four-byte input word and
      // applies the S-box to each of the four bytes to produce an output word.

      // Function Subword()
      {
        tempa[0] = getSBoxValue(tempa[0]);
        tempa[1] = getSBoxValue(tempa[1]);
        tempa[2] = getSBoxValue(tempa[2]);
        tempa[3] = getSBoxValue(tempa[3]);
      }
      tempa[0] = tempa[0] ^ Rcon[i / Nk];
    }
#if defined(AES256) && (AES256 == 1)
    if (i % Nk == 4)
    {
      // Function Subword()
      {
        tempa[0] = getSBoxValue(tempa[0]);
        tempa[1] = getSBoxValue(tempa[1]);
        tempa[2] = getSBoxValue(tempa[2]);
        tempa[3] = getSBoxValue(tempa[3]);
      }
    }
#endif
    j = i * 4;
    k = (i - Nk) * 4;
    RoundKey[j + 0] = RoundKey[k + 0] ^ tempa[0];
    RoundKey[j + 1] = RoundKey[k + 1] ^ tempa[1];
    RoundKey[j + 2] = RoundKey[k + 2] ^ tempa[2];
    RoundKey[j + 3] = RoundKey[k + 3] ^ tempa[3];
  }
}

// This function adds the round key to state.
// The round key is added to the state by an XOR function.
/* OPT: it makes sense to keep the round keys in memory because there are
only four round keys so at least 4 threads in half the warp (16 threads) 
will access the same key */
TORCH_CSPRNG_HOST_DEVICE void AddRoundKey(uint8_t round, state_t *state, const uint8_t *RoundKey)
{
  uint8_t i, j;
  for (i = 0; i < 4; ++i)
  {
    for (j = 0; j < 4; ++j)
    {
      (*state)[i][j] ^= RoundKey[(round * Nb * 4) + (i * Nb) + j];
    }
  }
}

// The SubBytes Function Substitutes the values in the
// state matrix with values in an S-box.
/* OPT: might want to bind this to texture instead of constant memory */
TORCH_CSPRNG_HOST_DEVICE void SubBytes(state_t *state)
{
  uint8_t i, j;
  for (i = 0; i < 4; ++i)
  {
    for (j = 0; j < 4; ++j)
    {
      (*state)[j][i] = getSBoxValue((*state)[j][i]);
    }
  }
}

// The ShiftRows() function shifts the rows in the state to the left.
// Each row is shifted with different offset.
// Offset = Row number. So the first row is not shifted.
/* OPT: can use a hardware intrinsic for the permutation */
TORCH_CSPRNG_HOST_DEVICE void ShiftRows(state_t *state)
{
  uint8_t temp;

  // Rotate first row 1 columns to left
  temp = (*state)[0][1];
  (*state)[0][1] = (*state)[1][1];
  (*state)[1][1] = (*state)[2][1];
  (*state)[2][1] = (*state)[3][1];
  (*state)[3][1] = temp;

  // Rotate second row 2 columns to left
  temp = (*state)[0][2];
  (*state)[0][2] = (*state)[2][2];
  (*state)[2][2] = temp;

  temp = (*state)[1][2];
  (*state)[1][2] = (*state)[3][2];
  (*state)[3][2] = temp;

  // Rotate third row 3 columns to left
  temp = (*state)[0][3];
  (*state)[0][3] = (*state)[3][3];
  (*state)[3][3] = (*state)[2][3];
  (*state)[2][3] = (*state)[1][3];
  (*state)[1][3] = temp;
}

TORCH_CSPRNG_HOST_DEVICE uint8_t xtime(uint8_t x)
{
  return ((x << 1) ^ (((x >> 7) & 1) * 0x1b));
}

// MixColumns function mixes the columns of the state matrix
TORCH_CSPRNG_HOST_DEVICE void MixColumns(state_t *state)
{
  uint8_t i;
  uint8_t Tmp, Tm, t;
  for (i = 0; i < 4; ++i)
  {
    t = (*state)[i][0];
    Tmp = (*state)[i][0] ^ (*state)[i][1] ^ (*state)[i][2] ^ (*state)[i][3];
    Tm = (*state)[i][0] ^ (*state)[i][1];
    Tm = xtime(Tm);
    (*state)[i][0] ^= Tm ^ Tmp;
    Tm = (*state)[i][1] ^ (*state)[i][2];
    Tm = xtime(Tm);
    (*state)[i][1] ^= Tm ^ Tmp;
    Tm = (*state)[i][2] ^ (*state)[i][3];
    Tm = xtime(Tm);
    (*state)[i][2] ^= Tm ^ Tmp;
    Tm = (*state)[i][3] ^ t;
    Tm = xtime(Tm);
    (*state)[i][3] ^= Tm ^ Tmp;
  }
}

TORCH_CSPRNG_HOST_DEVICE uint8_t Multiply(uint8_t x, uint8_t y)
{
  return (((y & 1) * x) ^
          ((y >> 1 & 1) * xtime(x)) ^
          ((y >> 2 & 1) * xtime(xtime(x))) ^
          ((y >> 3 & 1) * xtime(xtime(xtime(x)))) ^
          ((y >> 4 & 1) * xtime(xtime(xtime(xtime(x)))))); /* this last call to xtime() can be omitted */
}

// MixColumns function mixes the columns of the state matrix.
// The method used to multiply may be difficult to understand for the inexperienced.
// Please use the references to gain more information.
TORCH_CSPRNG_HOST_DEVICE void InvMixColumns(state_t *state)
{
  int i;
  uint8_t a, b, c, d;
  for (i = 0; i < 4; ++i)
  {
    a = (*state)[i][0];
    b = (*state)[i][1];
    c = (*state)[i][2];
    d = (*state)[i][3];

    (*state)[i][0] = Multiply(a, 0x0e) ^ Multiply(b, 0x0b) ^ Multiply(c, 0x0d) ^ Multiply(d, 0x09);
    (*state)[i][1] = Multiply(a, 0x09) ^ Multiply(b, 0x0e) ^ Multiply(c, 0x0b) ^ Multiply(d, 0x0d);
    (*state)[i][2] = Multiply(a, 0x0d) ^ Multiply(b, 0x09) ^ Multiply(c, 0x0e) ^ Multiply(d, 0x0b);
    (*state)[i][3] = Multiply(a, 0x0b) ^ Multiply(b, 0x0d) ^ Multiply(c, 0x09) ^ Multiply(d, 0x0e);
  }
}

// The SubBytes Function Substitutes the values in the
// state matrix with values in an S-box.
TORCH_CSPRNG_HOST_DEVICE void InvSubBytes(state_t *state)
{
  uint8_t i, j;
  for (i = 0; i < 4; ++i)
  {
    for (j = 0; j < 4; ++j)
    {
      (*state)[j][i] = getSBoxInvert((*state)[j][i]);
    }
  }
}

TORCH_CSPRNG_HOST_DEVICE void InvShiftRows(state_t *state)
{
  uint8_t temp;

  // Rotate first row 1 columns to right
  temp = (*state)[3][1];
  (*state)[3][1] = (*state)[2][1];
  (*state)[2][1] = (*state)[1][1];
  (*state)[1][1] = (*state)[0][1];
  (*state)[0][1] = temp;

  // Rotate second row 2 columns to right
  temp = (*state)[0][2];
  (*state)[0][2] = (*state)[2][2];
  (*state)[2][2] = temp;

  temp = (*state)[1][2];
  (*state)[1][2] = (*state)[3][2];
  (*state)[3][2] = temp;

  // Rotate third row 3 columns to right
  temp = (*state)[0][3];
  (*state)[0][3] = (*state)[1][3];
  (*state)[1][3] = (*state)[2][3];
  (*state)[2][3] = (*state)[3][3];
  (*state)[3][3] = temp;
}

// void populateRoundKeys()
// {
//   uint8_t key[16];
//   memset(key, 0, 16);
//   for (uint8_t i = 0; i < 4; i++)
//   {
//     key[0] = i;
//     KeyExpansion(h_RoundKeys[i], key);
//   }
// }

__host__ __device__ void printAESBlock2(uint8_t *b)
{
    for (int i = 0; i < 16; i+=4)
        printf("%02x%02x%02x%02x ", b[i], b[i+1], b[i+2], b[i+3]);
    printf("\n");
}

TORCH_CSPRNG_HOST_DEVICE void aes_encrypt(uint8_t *RoundKey, uint8_t *state /*plaintext*/)
{
  uint8_t round = 0;
  // Add the First round key to the state before starting the rounds.
  AddRoundKey(0, (state_t *)state, RoundKey);
  // printAESBlock2(state);

  // There will be Nr rounds.
  // The first Nr-1 rounds are identical.
  // These Nr rounds are executed in the loop below.
  // Last one without MixColumns()
  for (round = 1;; ++round)
  {
    SubBytes((state_t *)state);
    ShiftRows((state_t *)state);
    if (round == Nr)
    {
      break;
    }
    MixColumns((state_t *)state);
    AddRoundKey(round, (state_t *)state, RoundKey);
    // printAESBlock2(state);

  }
  // Add round key to last round
  AddRoundKey(Nr, (state_t *)state, RoundKey);
  // printAESBlock2(state);
}



TORCH_CSPRNG_HOST_DEVICE void apply_aes_prg(const uint8_t *key, int pt, uint8_t *ct1, uint8_t *ct2)
{
  uint8_t RoundKey[176];
  KeyExpansion(RoundKey, key);
  // printAESBlock2((uint8_t*) &RoundKey[16]);

  memset(ct1, 0, 16);
  memset(ct2, 0, 16);
  // ct1[0] = ((uint8_t*) pt)[3];
  // ct2[0] = ct1[0] + 2;
  *ct1 = pt;
  *ct2 = pt + 2;
  // printf("ciphertext\n");
  // printAESBlock2((uint8_t*) ct1);
  // printAESBlock2((uint8_t*) ct2);
  // printf("0\n");
  aes_encrypt(RoundKey, ct1);
  // printf("2\n");
  aes_encrypt(RoundKey, ct2);
}

__global__ void apply_aes_prg_wrapper(const uint8_t *key, int pt, uint8_t *ct1, uint8_t *ct2, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
  {
    uint8_t local_key[16];
    uint8_t local_ct1[16];
    uint8_t local_ct2[16];

    memcpy(local_key, &key[16 * i], 16);

    for(int j = 0; j < 64; j++)
    {
      memset(local_ct1, 0, 16);
      memset(local_ct2, 0, 16);

      apply_aes_prg(local_key, pt, local_ct1, local_ct2);
      // printAESBlock2(local_ct1);
      memcpy(local_key, local_ct1, 16);
    }
    memcpy(&ct1[16 * i], local_ct1, 16);
    memcpy(&ct2[16 * i], local_ct2, 16);
  }
}

extern "C" void test_prg(const uint8_t *key, int pt, uint8_t *ct1, uint8_t *ct2, int num_blocks, int tb_size)
{
  uint8_t *d_key, *d_ct1, *d_ct2;
  int n = num_blocks * tb_size;
  size_t mem_size = n * sizeof(uint8_t) * 16;

  checkCudaErrors(hipMalloc(&d_key, mem_size));
  checkCudaErrors(hipMalloc(&d_ct1, mem_size));
  checkCudaErrors(hipMalloc(&d_ct2, mem_size));

  hipMemcpy(d_key, key, mem_size, hipMemcpyHostToDevice);

  apply_aes_prg_wrapper<<<num_blocks, tb_size>>>((const uint8_t *)d_key, pt, (uint8_t *)d_ct1, (uint8_t *)d_ct2, n);

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipMemcpy(ct1, d_ct1, mem_size, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(ct2, d_ct2, mem_size, hipMemcpyDeviceToHost));
}
