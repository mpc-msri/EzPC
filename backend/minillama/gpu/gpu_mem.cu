#include <chrono>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include "hip/hip_runtime_api.h"
#include "gpu_stats.h"

// #include <sys/types.h>

extern "C" uint8_t* gpuMalloc(size_t size_in_bytes) {
    uint8_t* d_a;
    checkCudaErrors(hipMalloc(&d_a, size_in_bytes));
    return d_a;
}

extern "C" uint8_t* cpuMalloc(size_t size_in_bytes) {
    uint8_t* h_a;
    checkCudaErrors(hipHostMalloc(&h_a, size_in_bytes));
    return h_a;
}

extern "C" void gpuFree(void* d_a) {
    checkCudaErrors(hipFree(d_a));
}


extern "C" void cpuFree(void* h_a) {
    checkCudaErrors(hipHostFree(h_a));
}


extern "C" uint8_t* moveToCPU(uint8_t* d_a, size_t size_in_bytes, Stats* s) {
    uint8_t* h_a = cpuMalloc(size_in_bytes);
    auto start = std::chrono::high_resolution_clock::now();  
    checkCudaErrors(hipMemcpy(h_a, d_a, size_in_bytes, hipMemcpyDeviceToHost));
    auto end = std::chrono::high_resolution_clock::now();  
    auto elapsed = end - start;
    if(s) s->transfer_time += std::chrono::duration_cast<std::chrono::milliseconds>(elapsed).count();
    // cout << "move to cpu " << s->transfer_time << endl;
    return h_a;
}

extern "C" uint8_t* moveIntoCPUMem(uint8_t* h_a, uint8_t* d_a, size_t size_in_bytes, Stats* s) {
    // uint8_t* h_a = cpuMalloc(size_in_bytes);
    auto start = std::chrono::high_resolution_clock::now();  
    checkCudaErrors(hipMemcpy(h_a, d_a, size_in_bytes, hipMemcpyDeviceToHost));
    auto end = std::chrono::high_resolution_clock::now();  
    auto elapsed = end - start;
    if(s) s->transfer_time += std::chrono::duration_cast<std::chrono::milliseconds>(elapsed).count();
    // cout << "move to cpu " << s->transfer_time << endl;
    return h_a;
}

extern "C" uint8_t* moveToGPU(uint8_t* h_a, size_t size_in_bytes, Stats* s) {
    uint8_t* d_a = gpuMalloc(size_in_bytes);
    auto start = std::chrono::high_resolution_clock::now();  
    checkCudaErrors(hipMemcpy(d_a, h_a, size_in_bytes, hipMemcpyHostToDevice));
    auto end = std::chrono::high_resolution_clock::now();  
    auto elapsed = end - start;
    if(s) s->transfer_time += std::chrono::duration_cast<std::chrono::milliseconds>(elapsed).count();
    return d_a;
}
