// Author: Neha Jawalkar
// Copyright:
// 
// Copyright (c) 2024 Microsoft Research
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <chrono>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include "hip/hip_runtime_api.h"
#include "gpu_stats.h"
#include <cassert>

// #include <sys/types.h>

hipMemPool_t mempool;

extern "C" void initGPUMemPool()
{
    int isMemPoolSupported = 0;
    int device = 0;
    // is it okay to use device=0?
    checkCudaErrors(hipDeviceGetAttribute(&isMemPoolSupported,
                                           hipDeviceAttributeMemoryPoolsSupported, device));
    // printf("%d\n", isMemPoolSupported);
    assert(isMemPoolSupported);
    /* implicitly assumes that the device is 0 */

    checkCudaErrors(hipDeviceGetDefaultMemPool(&mempool, device));
    uint64_t threshold = UINT64_MAX;
    checkCudaErrors(hipMemPoolSetAttribute(mempool, hipMemPoolAttrReleaseThreshold, &threshold));
    uint64_t *d_dummy_ptr;
    uint64_t bytes = 40 * (1ULL << 30);
    checkCudaErrors(hipMallocAsync(&d_dummy_ptr, bytes, 0));
    checkCudaErrors(hipFreeAsync(d_dummy_ptr, 0));
    uint64_t reserved_read, threshold_read;
    checkCudaErrors(hipMemPoolGetAttribute(mempool, hipMemPoolAttrReservedMemCurrent, &reserved_read));
    checkCudaErrors(hipMemPoolGetAttribute(mempool, hipMemPoolAttrReleaseThreshold, &threshold_read));
    printf("reserved memory: %lu %lu\n", reserved_read, threshold_read);
}

extern "C" uint8_t *gpuMalloc(size_t size_in_bytes)
{
    uint8_t *d_a;
    checkCudaErrors(hipMallocAsync(&d_a, size_in_bytes, 0));
    return d_a;
}


extern "C" uint8_t *cpuMalloc(size_t size_in_bytes, bool pin)
{
    uint8_t *h_a;
    int err = posix_memalign((void **)&h_a, 32, size_in_bytes);
    assert(err == 0 && "posix memalign");
    if (pin)
        checkCudaErrors(hipHostRegister(h_a, size_in_bytes, hipHostRegisterDefault));
    return h_a;
}

extern "C" void gpuFree(void *d_a)
{
    checkCudaErrors(hipFreeAsync(d_a, 0));
}

extern "C" void cpuFree(void *h_a, bool pinned)
{
    if (pinned)
        checkCudaErrors(hipHostUnregister(h_a));
    free(h_a);
}

extern "C" uint8_t *moveToCPU(uint8_t *d_a, size_t size_in_bytes, Stats *s)
{
    uint8_t *h_a = cpuMalloc(size_in_bytes, true);
    auto start = std::chrono::high_resolution_clock::now();
    checkCudaErrors(hipMemcpy(h_a, d_a, size_in_bytes, hipMemcpyDeviceToHost));
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;
    if (s)
        s->transfer_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
    return h_a;
}

extern "C" uint8_t *moveIntoGPUMem(uint8_t *d_a, uint8_t *h_a, size_t size_in_bytes, Stats *s)
{
    auto start = std::chrono::high_resolution_clock::now();
    checkCudaErrors(hipMemcpy(d_a, h_a, size_in_bytes, hipMemcpyHostToDevice));
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;
    if (s)
        s->transfer_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
    return h_a;
}

extern "C" uint8_t *moveIntoCPUMem(uint8_t *h_a, uint8_t *d_a, size_t size_in_bytes, Stats *s)
{
    auto start = std::chrono::high_resolution_clock::now();
    checkCudaErrors(hipMemcpy(h_a, d_a, size_in_bytes, hipMemcpyDeviceToHost));
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;
    if (s)
        s->transfer_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
    return h_a;
}

extern "C" uint8_t *moveToGPU(uint8_t *h_a, size_t size_in_bytes, Stats *s)
{
    uint8_t *d_a = gpuMalloc(size_in_bytes);
    auto start = std::chrono::high_resolution_clock::now();
    checkCudaErrors(hipMemcpy(d_a, h_a, size_in_bytes, hipMemcpyHostToDevice));
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;
    if (s)
        s->transfer_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
    return d_a;
}
