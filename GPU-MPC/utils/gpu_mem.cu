#include <chrono>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include "hip/hip_runtime_api.h"
#include "gpu_stats.h"
#include <cassert>

// #include <sys/types.h>

hipMemPool_t mempool;

extern "C" void initGPUMemPool()
{
    int isMemPoolSupported = 0;
    int device = 0;
    // is it okay to use device=0?
    checkCudaErrors(hipDeviceGetAttribute(&isMemPoolSupported,
                                           hipDeviceAttributeMemoryPoolsSupported, device));
    // printf("%d\n", isMemPoolSupported);
    assert(isMemPoolSupported);
    /* implicitly assumes that the device is 0 */

    checkCudaErrors(hipDeviceGetDefaultMemPool(&mempool, device));
    uint64_t threshold = UINT64_MAX;
    checkCudaErrors(hipMemPoolSetAttribute(mempool, hipMemPoolAttrReleaseThreshold, &threshold));
    uint64_t *d_dummy_ptr;
    uint64_t bytes = 40 * (1ULL << 30);
    checkCudaErrors(hipMallocAsync(&d_dummy_ptr, bytes, 0));
    checkCudaErrors(hipFreeAsync(d_dummy_ptr, 0));
    uint64_t reserved_read, threshold_read;
    checkCudaErrors(hipMemPoolGetAttribute(mempool, hipMemPoolAttrReservedMemCurrent, &reserved_read));
    checkCudaErrors(hipMemPoolGetAttribute(mempool, hipMemPoolAttrReleaseThreshold, &threshold_read));
    printf("reserved memory: %lu %lu\n", reserved_read, threshold_read);
}

extern "C" uint8_t *gpuMalloc(size_t size_in_bytes)
{
    uint8_t *d_a;
    checkCudaErrors(hipMallocAsync(&d_a, size_in_bytes, 0));
    return d_a;
}


extern "C" uint8_t *cpuMalloc(size_t size_in_bytes, bool pin)
{
    uint8_t *h_a;
    int err = posix_memalign((void **)&h_a, 32, size_in_bytes);
    assert(err == 0 && "posix memalign");
    if (pin)
        checkCudaErrors(hipHostRegister(h_a, size_in_bytes, hipHostRegisterDefault));
    return h_a;
}

extern "C" void gpuFree(void *d_a)
{
    checkCudaErrors(hipFreeAsync(d_a, 0));
}

extern "C" void cpuFree(void *h_a, bool pinned)
{
    if (pinned)
        checkCudaErrors(hipHostUnregister(h_a));
    free(h_a);
}

extern "C" uint8_t *moveToCPU(uint8_t *d_a, size_t size_in_bytes, Stats *s)
{
    uint8_t *h_a = cpuMalloc(size_in_bytes, true);
    auto start = std::chrono::high_resolution_clock::now();
    checkCudaErrors(hipMemcpy(h_a, d_a, size_in_bytes, hipMemcpyDeviceToHost));
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;
    if (s)
        s->transfer_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
    return h_a;
}

extern "C" uint8_t *moveIntoGPUMem(uint8_t *d_a, uint8_t *h_a, size_t size_in_bytes, Stats *s)
{
    auto start = std::chrono::high_resolution_clock::now();
    checkCudaErrors(hipMemcpy(d_a, h_a, size_in_bytes, hipMemcpyHostToDevice));
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;
    if (s)
        s->transfer_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
    return h_a;
}

extern "C" uint8_t *moveIntoCPUMem(uint8_t *h_a, uint8_t *d_a, size_t size_in_bytes, Stats *s)
{
    auto start = std::chrono::high_resolution_clock::now();
    checkCudaErrors(hipMemcpy(h_a, d_a, size_in_bytes, hipMemcpyDeviceToHost));
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;
    if (s)
        s->transfer_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
    return h_a;
}

extern "C" uint8_t *moveToGPU(uint8_t *h_a, size_t size_in_bytes, Stats *s)
{
    uint8_t *d_a = gpuMalloc(size_in_bytes);
    auto start = std::chrono::high_resolution_clock::now();
    checkCudaErrors(hipMemcpy(d_a, h_a, size_in_bytes, hipMemcpyHostToDevice));
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;
    if (s)
        s->transfer_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
    return d_a;
}
