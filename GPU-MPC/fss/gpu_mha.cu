#include "hip/hip_runtime.h"
// 
// Copyright:
// 
// Copyright (c) 2024 Microsoft Research
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <cstddef>
#include <cstdint>
#include <cassert>
#include <cmath>

#include "gpu_mha.h"

#include "utils/gpu_mem.h"
#include "utils/gpu_file_utils.h"
#include "utils/misc_utils.h"
#include "utils/gpu_random.h"

#include "fss/gpu_scalarmul.h"
#include "fss/gpu_truncate.h"


template <typename T>
__global__ void rotEmbKernel(MHAParams pMHA, int scale, u64 N, T *X, T *Y)
{
    // the vectors are N x dim_W
    assert(pMHA.dim_W % 2 == 0);
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // int N = n_seq * dim_W;
    int dim_W_half = pMHA.dim_W / 2;
    if (tid < N)
    {
        // n_seq * dim_W * n_heads
        int temp = tid;
        int head = tid / (pMHA.n_seq * pMHA.dim_W);
        temp = temp % (pMHA.n_seq * pMHA.dim_W);
        int i = temp / pMHA.dim_W;
        int j = temp % pMHA.dim_W;
        float sinx, cosx;
        auto k = j - (j >= dim_W_half) * dim_W_half;
        __sincosf(i / __powf(10000, (2 * k / (float)pMHA.dim_W)), &sinx, &cosx);
        const auto uLim = T(1ULL << (scale - 3));
        T sinxi = T(sinx * uLim);
        T cosxi = T(cosx * uLim);
        if (sinxi == uLim)
            sinxi -= 1;
        if (cosxi == uLim)
            cosxi -= 1;

        auto l = (j + dim_W_half) % pMHA.dim_W;
        T m1 = 2 * (j >= dim_W_half) - 1;
        Y[tid] = cosxi * X[tid] + m1 * sinxi * X[head * pMHA.n_seq * pMHA.dim_W + i * pMHA.dim_W + l];
    }
}

template <typename T>
T *gpuKeygenRotEmb(u8 **key_as_bytes, int party, int bw, int scale, MHAParams pMHA, T *d_mask_X, AESGlobalContext *g)
{
    printf("*********** Generating rotary embedding key! ***************\n");
    size_t size_X = pMHA.n_heads * (u64)pMHA.n_seq * pMHA.dim_W;
    auto d_mask_X1 = (T *)gpuMalloc(size_X * sizeof(T));
    rotEmbKernel<<<(size_X - 1) / 128 + 1, 128>>>(pMHA, scale, size_X, d_mask_X, d_mask_X1);
    // gpuFree(d_mask_X);
    auto d_mask_truncated_X = genGPUTruncateKey<T, T>(key_as_bytes, party, TruncateType::TrWithSlack, bw, bw, scale - 3, size_X, d_mask_X1, g);
    gpuFree(d_mask_X1);
    return d_mask_truncated_X;
}

template <typename T>
T *gpuRotEmb(SigmaPeer *peer, int party, int bw, int scale, MHAParams pMHA, GPUTruncateKey<T> trKey, T *d_X, AESGlobalContext *g, Stats *s)
{
    u64 b0 = peer->bytesSent() + peer->bytesReceived();

    size_t size_X = pMHA.n_heads * (u64)pMHA.n_seq * pMHA.dim_W;
    auto d_X1 = (T *)gpuMalloc(size_X * sizeof(T));
    rotEmbKernel<<<(size_X - 1) / 128 + 1, 128>>>(pMHA, scale, size_X, d_X, d_X1);
    // don't free this because QKV is one long array
    // gpuFree(d_X);
    auto d_truncated_X = gpuTruncate<T, T>(bw, bw, TruncateType::TrWithSlack, trKey, scale - 3, peer, party, size_X, d_X1, g, s); //, true);
    gpuFree(d_X1);

    u64 b1 = peer->bytesSent() + peer->bytesReceived();
    s->linear_comm_bytes += (b1 - b0);
    printf("Comm in rotary embedding=%lu, N=%d\n", b1 - b0, size_X);
    return d_truncated_X;
}

// neha: to fix: maxpool, and make it so the conv2d output is 40 bits???? (bout == 40????)
template <typename T>
T *gpuKeygenMHA(u8 **key_as_bytes, int party, int bw, int scale, MHAParams pMHA, MHAMulParams pMHAMul, T *WQKV, T *YQKV, T *WProj, T *YProj, T *d_mask_X, AESGlobalContext *g)
{
    auto d_mask_QKV = gpuKeygenMatmul(key_as_bytes, party, pMHAMul.pQKV, d_mask_X, WQKV, YQKV, TruncateType::TrFloor, g);
    // this->activation.d_data = d_mask_QKV;

    // printf("Size of Q=%d\n", pQKV.size_C / 3);
    int QKSz = pMHAMul.pQKV.size_C / 3;
    auto d_mask_Q = d_mask_QKV;
    auto d_mask_K = d_mask_QKV + QKSz;
    auto d_mask_V = d_mask_K + QKSz;
    // this->activation.d_data = d_mask_V;
    if (pMHA.rotEmb)
    {
        d_mask_Q = gpuKeygenRotEmb(key_as_bytes, party, bw, scale, pMHA, d_mask_Q, g);
        d_mask_K = gpuKeygenRotEmb(key_as_bytes, party, bw, scale, pMHA, d_mask_K, g);
    }

    auto d_mask_QKt = gpuKeygenMatmul(key_as_bytes, party, pMHAMul.pQKt, d_mask_Q, d_mask_K, (T *)NULL, TruncateType::TrFloor, g, true);
    if (pMHA.rotEmb)
    {
        gpuFree(d_mask_Q);
        gpuFree(d_mask_K);
    }

    T *d_mask_normQKt = d_mask_QKt;
    if (pMHA.doNormQKt && int(log2(pMHA.dim_W)) % 2 == 1)
    {
        T invSqrtDimW = T((1.0f / sqrt(double(pMHA.dim_W))) * (1LL << scale));
        printf("####### Doing a scalar multiplication\n");
        d_mask_normQKt = gpuKeygenScalarMul(key_as_bytes, party, bw, pMHAMul.pQKt.size_C, invSqrtDimW, d_mask_QKt, TruncateType::TrFloor, scale, g);
        gpuFree(d_mask_QKt);
    }

    auto d_mask_smQKt = gpuKeygenSoftmax(key_as_bytes, party, pMHAMul.pMPool, d_mask_normQKt, g);
    gpuFree(d_mask_normQKt);
    // this->activation.d_data = d_mask_smQKt;
    auto d_mask_smQKtV = gpuKeygenMatmul(key_as_bytes, party, pMHAMul.pSmQKtV, d_mask_smQKt, d_mask_V, (T *)NULL, TruncateType::TrFloor, g, true);
    gpuFree(d_mask_smQKt);
    gpuFree(d_mask_QKV);
    // this->activation.d_data = d_mask_smQKtV;

    auto d_mask_proj = gpuKeygenMatmul(key_as_bytes, party, pMHAMul.pProj, d_mask_smQKtV, WProj, YProj, TruncateType::TrFloor, g);
    gpuFree(d_mask_smQKtV);
    // free gpu memory
    // this->activation.d_data = d_mask_proj;
    return d_mask_proj;
}

template <typename T>
T *gpuMHA(SigmaPeer *peer, int party, int bw, int scale, MHAParams pMHA, MHAMulParams pMHAMul, GPUMHAKey<T> k, T *WQKV, T *YQKV, T *WProj, T *YProj, T *d_X, MHATables<T> t, AESGlobalContext *g, Stats *s)
{
    auto b0 = peer->bytesSent() + peer->bytesReceived();

    auto d_QKV = gpuMatmul(peer, party, pMHAMul.pQKV, k.mmKeyQKV, d_X, WQKV, YQKV, TruncateType::TrFloor, g, s);
    // this->activation.d_data = d_QKV;
    size_t QKSz = pMHAMul.pQKV.size_C / 3;
    auto d_Q = d_QKV;
    auto d_K = d_QKV + QKSz;
    auto d_V = d_K + QKSz;
    // this->activation.d_data = d_V;

    if (pMHA.rotEmb)
    {
        d_Q = gpuRotEmb(peer, party, bw, scale, pMHA, k.reQTrKey, d_Q, g, s);
        d_K = gpuRotEmb(peer, party, bw, scale, pMHA, k.reKTrKey, d_K, g, s);
    }

    auto d_QKt = gpuMatmul(peer, party, pMHAMul.pQKt, k.mmKeyQKt, d_Q, d_K, (T *)NULL, TruncateType::TrFloor, g, s, true);
    if (pMHA.rotEmb)
    {
        gpuFree(d_Q);
        gpuFree(d_K);
    }
    // this->activation.d_data = d_QKt;

    T *d_normQKt = d_QKt;

    if (pMHA.doNormQKt && int(log2(pMHA.dim_W)) % 2 == 1)
    {
        T invSqrtDimW = T((1.0f / sqrt(double(pMHA.dim_W))) * (1LL << scale));
        printf("####### Doing a scalar multiplication\n");
        d_normQKt = gpuScalarMul(peer, party, bw, pMHAMul.pQKt.size_C, k.normQKtTrKey, invSqrtDimW, d_QKt, TruncateType::TrFloor, scale, g, s);
        gpuFree(d_QKt);
    }

    // assert(d_nExpMsbTab);
    // assert(d_nExpLsbTab);
    // assert(d_invTab);
    // this->activation.d_data = d_normQKt;

    auto d_smQKt = gpuSoftmax(peer, party, pMHAMul.pMPool, k.softmaxKey, d_normQKt, t.d_nExpMsbTab, t.d_nExpLsbTab, t.d_invTab, g, s);
    gpuFree(d_normQKt);
    // this->activation.d_data = d_smQKt;
    auto d_smQKtV = gpuMatmul(peer, party, pMHAMul.pSmQKtV, k.mmKeySmQKtV, d_smQKt, d_V, (T *)NULL, TruncateType::TrFloor, g, s, true);
    gpuFree(d_smQKt);
    gpuFree(d_QKV);
    // // this->activation.d_data = d_smQKtV;
    auto d_proj = gpuMatmul(peer, party, pMHAMul.pProj, k.mmKeyProj, d_smQKtV, WProj, YProj, TruncateType::TrFloor, g, s);
    gpuFree(d_smQKtV);
    auto b1 = peer->bytesSent() + peer->bytesReceived();
    printf("MHA Comm=%ld\n", b1 - b0);
    return d_proj;
}
