#include "utils/gpu_random.h"
#include "gpu_dcf.h"
#include "gpu_maxpool.h"

namespace dcf
{
    template <typename T>
    T *gpuMaxpoolHelper(SigmaPeer *peer, int party, MaxpoolParams p, GPU2RoundReLUKey<T> k, GPUAndKey andKey, int i, int j, T *d_I, T *d_curMax, u32 *d_oneHot, AESGlobalContext *gaes, Stats *s)
    {
        int outSz = getMSz(p);
        T *d_diff = (T *)gpuMalloc(outSz * sizeof(T));
        diffWithCurMax<<<(outSz - 1) / 256 + 1, 256>>>(p, i, j, d_curMax, d_I, d_diff, outSz);
        checkCudaErrors(hipDeviceSynchronize());
        auto d_res = gpuTwoRoundRelu(peer, party, k, d_diff, gaes, s);
        auto d_drelu = d_res.first;
        auto d_newMax = d_res.second;
        gpuFree(d_diff);
        // relu(x-y) + y
        gpuLinearComb(p.bw, outSz, d_newMax, T(1), d_newMax, T(1), d_curMax);
        if (d_oneHot)
        {
            gpuAndForMaxpool(p, i * p.FW + j + 1, andKey, d_drelu, d_oneHot, party, s);
            int numBits = k.selectKey.N * p.FH * p.FW;
            peer->reconstructInPlace((T *)d_oneHot, 1, numBits, s);
        }
        gpuFree(d_drelu);
        return d_newMax;
    }

    template <typename T>
    T *gpuMaxPool(SigmaPeer *peer, int party, MaxpoolParams p, GPUMaxpoolKey<T> k, T *d_I, u32 *d_oneHot,
                  AESGlobalContext *gaes, Stats *s)

    {
        int outSz = getMSz(p);
        T *d_curMax = (T *)gpuMalloc(outSz * sizeof(T));
        populateCurMax<<<(outSz - 1) / 256 + 1, 256>>>(p, d_curMax, d_I, outSz);
        checkCudaErrors(hipDeviceSynchronize());
        for (int i = 0; i < p.FH; i++)
        {
            for (int j = 0; j < p.FW; j++)
            {
                if (i == 0 && j == 0)
                    continue;
                auto d_newMax = gpuMaxpoolHelper(peer, party, p, k.reluKey[i * p.FW + j], k.andKey[i * p.FW + j], i, j, d_I, d_curMax, d_oneHot, gaes, s);
                gpuFree(d_curMax);
                d_curMax = d_newMax;
            }
        }
        return d_curMax;
    }

    template <typename T>
    T *gpuKeygenMaxpoolHelper(uint8_t **key_as_bytes, int party, MaxpoolParams p, int fh, int fw,
                              T *d_inputMask, T *d_curMaxMask, u8 *d_oneHotMask,
                              AESGlobalContext *gaes)
    {
        int outSz = getMSz(p);
        T *d_diffMask = (T *)gpuMalloc(outSz * sizeof(T));
        // d_diffMask = inputMask - curMask
        diffWithCurMax<<<(outSz - 1) / 256 + 1, 256>>>(p, fh, fw, d_curMaxMask, d_inputMask, d_diffMask, outSz);
        checkCudaErrors(hipDeviceSynchronize());
        auto d_res = gpuGenTwoRoundReluKey(key_as_bytes, party, p.bin, p.bw, outSz, d_diffMask, gaes);
        auto d_dreluMask = d_res.first;
        auto d_newMaxMask = d_res.second;
        gpuFree(d_diffMask);
        gpuLinearComb(p.bw, outSz, d_newMaxMask, T(1), d_newMaxMask, T(1), d_curMaxMask);
        if (d_oneHotMask)
        {
            gpuKeygenOneHotMaxpool(key_as_bytes, party, p, outSz, fh, fw, d_dreluMask, d_oneHotMask);
        }
        gpuFree(d_dreluMask);
        return d_newMaxMask;
    }

    template <typename T>
    T *gpuKeygenMaxpool(uint8_t **key_as_bytes, int party, MaxpoolParams p,
                        T *d_inputMask, u8 *d_oneHotMask,
                        AESGlobalContext *gaes)
    {
        int outSz = getMSz(p);
        T *d_curMaxMask = (T *)gpuMalloc(outSz * sizeof(T));
        populateCurMax<<<(outSz - 1) / 256 + 1, 256>>>(p, d_curMaxMask, d_inputMask, outSz);
        checkCudaErrors(hipDeviceSynchronize());
        for (int i = 0; i < p.FH; i++)
        {
            for (int j = 0; j < p.FW; j++)
            {
                if (i == 0 && j == 0)
                    continue;
                auto d_newMax = gpuKeygenMaxpoolHelper(key_as_bytes, party, p, i, j, d_inputMask, d_curMaxMask, d_oneHotMask, gaes);
                gpuFree(d_curMaxMask);
                d_curMaxMask = d_newMax;
            }
        }
        return d_curMaxMask;
    }
}