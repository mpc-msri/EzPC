#include "hip/hip_runtime.h"
// Author: Neha Jawalkar
// Copyright:
//
// Copyright (c) 2024 Microsoft Research
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// #pragma once

#include "utils/gpu_data_types.h"
#include "utils/hip/hip_runtime_api.h"
#include "utils/gpu_mem.h"

__device__ const u32 RCON32C[15] = {
	0x01000000, 0x02000000, 0x04000000, 0x08000000,
	0x10000000, 0x20000000, 0x40000000, 0x80000000,
	0x1B000000, 0x36000000, 0x6C000000, 0xD8000000,
	0xAB000000, 0x4D000000, 0x9A000000};

inline __device__ u32 readSBoxByte(u32 byteIn, u8 (*Sbox)[32][4])
{
	int wTid = threadIdx.x & 31;
	auto i = (byteIn & 0xff) / 4;
	return (u32)Sbox[i][wTid][byteIn & 3];
}

__device__ void aesKeySchedule(u32 *key, u32 *roundKey, u32 *t4_0S, u32 *t4_1S, u32 *t4_2S, u32 *t4_3S)
{
	u32 roundKey0, roundKey1, roundKey2, roundKey3;

	roundKey0 = key[0];
	roundKey1 = key[1];
	roundKey2 = key[2];
	roundKey3 = key[3];

	roundKey[0] = roundKey0;
	roundKey[1] = roundKey1;
	roundKey[2] = roundKey2;
	roundKey[3] = roundKey3;

	for (u8 r = 0; r < AES_128_ROUNDS; r++)
	{
		roundKey0 = roundKey0 ^ t4_3S[(roundKey3 >> 16) & 0xff] ^ t4_2S[(roundKey3 >> 8) & 0xff] ^ t4_1S[roundKey3 & 0xff] ^ t4_0S[(roundKey3 >> 24)] ^ RCON32C[r];
		roundKey1 = roundKey1 ^ roundKey0;
		roundKey2 = roundKey2 ^ roundKey1;
		roundKey3 = roundKey3 ^ roundKey2;

		roundKey[4 * r + 4] = roundKey0;
		roundKey[4 * r + 5] = roundKey1;
		roundKey[4 * r + 6] = roundKey2;
		roundKey[4 * r + 7] = roundKey3;
	}
}

inline __device__ u32 cyclicRot(u32 s, u32 rot)
{
	return __byte_perm(s, s, rot);
}

inline __device__ u32 computeOne(u32 s0, u32 s1, u32 s2, u32 s3, u32 *roundKey, int rkIdx, u32 (*t0_s)[NUM_SHARED_MEM_BANKS])
{
	int wTid = threadIdx.x & 31;
	return t0_s[__byte_perm(s0, 0, 0x4443)][wTid] ^
		   cyclicRot(t0_s[__byte_perm(s1, 0, 0x4442)][wTid], CYCLIC_ROT_RIGHT_1) ^
		   cyclicRot(t0_s[__byte_perm(s2, 0, 0x4441)][wTid], CYCLIC_ROT_RIGHT_2) ^
		   cyclicRot(t0_s[s3 & 0xff][wTid], CYCLIC_ROT_RIGHT_3) ^
		   roundKey[rkIdx];
}

inline __device__ u32 readSBoxByteAndCyclicShift(u32 byteIn, u8 (*Sbox)[32][4], int shift)
{
	return cyclicRot(readSBoxByte(byteIn, Sbox), shift);
}

inline __device__ u32 computeLast(u32 t0, u32 t1, u32 t2, u32 t3, u8 (*Sbox)[32][4], u32 roundKey)
{
	return readSBoxByteAndCyclicShift(t0 >> 24, Sbox, CYCLIC_ROT_RIGHT_1) ^ readSBoxByteAndCyclicShift(t1 >> 16, Sbox, CYCLIC_ROT_RIGHT_2) ^ readSBoxByteAndCyclicShift(t2 >> 8, Sbox, CYCLIC_ROT_RIGHT_3) ^ readSBoxByte(t3, Sbox) ^ roundKey;
}

__device__ void aesEncrypt(u32 *pt, u32 *roundKey, u32 (*t0_s)[NUM_SHARED_MEM_BANKS], u8 (*Sbox)[32][4])
{
	u32 s0, s1, s2, s3;
	s0 = pt[0];
	s1 = pt[1];
	s2 = pt[2];
	s3 = pt[3];
	s0 = s0 ^ roundKey[0];
	s1 = s1 ^ roundKey[1];
	s2 = s2 ^ roundKey[2];
	s3 = s3 ^ roundKey[3];

	u32 t0, t1, t2, t3;
	u32 rkIdx = 4;
	for (u8 r = 0; r < AES_128_ROUNDS_MIN_1; r++)
	{
		// Table based round function
		t0 = computeOne(s0, s1, s2, s3, roundKey, rkIdx, t0_s);
		t1 = computeOne(s1, s2, s3, s0, roundKey, rkIdx + 1, t0_s);
		t2 = computeOne(s2, s3, s0, s1, roundKey, rkIdx + 2, t0_s);
		t3 = computeOne(s3, s0, s1, s2, roundKey, rkIdx + 3, t0_s);
		s0 = t0;
		s1 = t1;
		s2 = t2;
		s3 = t3;
		rkIdx += 4;
	}
	s0 = computeLast(t0, t1, t2, t3, Sbox, roundKey[40]);
	s1 = computeLast(t1, t2, t3, t0, Sbox, roundKey[41]);
	s2 = computeLast(t2, t3, t0, t1, Sbox, roundKey[42]);
	s3 = computeLast(t3, t0, t1, t2, Sbox, roundKey[43]);

	pt[0] = s0;
	pt[1] = s1;
	pt[2] = s2;
	pt[3] = s3;
}

__device__ void loadSbox(AESGlobalContext *g, AESSharedContext *s)
{
	__shared__ u32 t0_s[AES_128_TABLE_SIZE][NUM_SHARED_MEM_BANKS];
	__shared__ u8 Sbox[64][32][4];
	__shared__ u32 t4_0S[AES_128_TABLE_SIZE];
	__shared__ u32 t4_1S[AES_128_TABLE_SIZE];
	__shared__ u32 t4_2S[AES_128_TABLE_SIZE];
	__shared__ u32 t4_3S[AES_128_TABLE_SIZE];
	// tb size might be small but it will be non-zero
	for (int i = 0; i < max(AES_128_TABLE_SIZE / blockDim.x, u32(1)); i++)
	{
		// stride
		int tid = threadIdx.x + i * blockDim.x;
		if (tid < AES_128_TABLE_SIZE)
		{
			t4_0S[tid] = g->t4_0G[tid];
			t4_1S[tid] = g->t4_1G[tid];
			t4_2S[tid] = g->t4_2G[tid];
			t4_3S[tid] = g->t4_3G[tid];
			for (u8 bank = 0; bank < NUM_SHARED_MEM_BANKS; bank++)
			{
				t0_s[tid][bank] = g->t0_g[tid];
				Sbox[tid / 4][bank][tid % 4] = g->Sbox_g[tid];
			}
		}
	}
	__syncthreads();
	s->t0_s = t0_s;
	s->Sbox = Sbox;
	s->t4_0S = t4_0S;
	s->t4_1S = t4_1S;
	s->t4_2S = t4_2S;
	s->t4_3S = t4_3S;
}

__device__ void reverseBytes(u32 *x)
{
	x[0] = __byte_perm(x[0], 0, 0x123);
	x[1] = __byte_perm(x[1], 0, 0x123);
	x[2] = __byte_perm(x[2], 0, 0x123);
	x[3] = __byte_perm(x[3], 0, 0x123);
}

__device__ void applyAESPRG(AESSharedContext *s, u32 *key, uint8_t pt, u32 *ct1)
{
	reverseBytes(key);
	u32 roundKey[44];
	aesKeySchedule(key, roundKey, s->t4_0S, s->t4_1S, s->t4_2S, s->t4_3S);
	memset(ct1, 0, 4 * sizeof(u32));
	((uint8_t *)ct1)[3] = pt;
	aesEncrypt(ct1, roundKey, s->t0_s, s->Sbox);
	reverseBytes(ct1);
}

__device__ void applyAESPRGTwoTimes(AESSharedContext *s, u32 *key, uint8_t pt, u32 *ct1, u32 *ct2)
{
	reverseBytes(key);
	u32 roundKey[44];
	aesKeySchedule(key, roundKey, s->t4_0S, s->t4_1S, s->t4_2S, s->t4_3S);
	memset(ct1, 0, 4 * sizeof(u32));
	memset(ct2, 0, 4 * sizeof(u32));
	((uint8_t *)ct1)[3] = pt;
	((uint8_t *)ct2)[3] = pt + 2;
	aesEncrypt(ct1, roundKey, s->t0_s, s->Sbox);
	aesEncrypt(ct2, roundKey, s->t0_s, s->Sbox);
	reverseBytes(ct1);
	reverseBytes(ct2);
}

__device__ void applyAESPRGFourTimes(AESSharedContext *s, u32 *key, u32 *ct1, u32 *ct2, u32 *ct3, u32 *ct4)
{
	reverseBytes(key);
	u32 roundKey[44];
	aesKeySchedule(key, roundKey, s->t4_0S, s->t4_1S, s->t4_2S, s->t4_3S);
	memset(ct1, 0, 4 * sizeof(u32));
	memset(ct2, 0, 4 * sizeof(u32));
	memset(ct3, 0, 4 * sizeof(u32));
	memset(ct4, 0, 4 * sizeof(u32));
	((uint8_t *)ct2)[3] = 1;
	((uint8_t *)ct3)[3] = 2;
	((uint8_t *)ct4)[3] = 3;
	aesEncrypt(ct1, roundKey, s->t0_s, s->Sbox);
	aesEncrypt(ct2, roundKey, s->t0_s, s->Sbox);
	aesEncrypt(ct3, roundKey, s->t0_s, s->Sbox);
	aesEncrypt(ct4, roundKey, s->t0_s, s->Sbox);
	reverseBytes(ct1);
	reverseBytes(ct2);
	reverseBytes(ct3);
	reverseBytes(ct4);
}

void initAESContext(AESGlobalContext *g)
{
	g->t0_g = (u32 *)moveToGPU((u8 *)T0, AES_128_TABLE_SIZE * sizeof(u32), NULL);
	g->Sbox_g = (u8 *)moveToGPU((u8 *)Sbox_g, 256 * sizeof(u8), NULL);
	g->t4_0G = (u32 *)moveToGPU((u8 *)T4_0, AES_128_TABLE_SIZE * sizeof(u32), NULL);
	g->t4_1G = (u32 *)moveToGPU((u8 *)T4_1, AES_128_TABLE_SIZE * sizeof(u32), NULL);
	g->t4_2G = (u32 *)moveToGPU((u8 *)T4_2, AES_128_TABLE_SIZE * sizeof(u32), NULL);
	g->t4_3G = (u32 *)moveToGPU((u8 *)T4_3, AES_128_TABLE_SIZE * sizeof(u32), NULL);
}

void freeAESGlobalContext(AESGlobalContext *g)
{
	gpuFree(g->t0_g);
	gpuFree(g->Sbox_g);
	gpuFree(g->t4_0G);
	gpuFree(g->t4_1G);
	gpuFree(g->t4_2G);
	gpuFree(g->t4_3G);
}
