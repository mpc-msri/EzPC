#include "utils/gpu_data_types.h"
#include "utils/gpu_file_utils.h"
#include "utils/misc_utils.h"
#include "utils/gpu_mem.h"

#include <cassert>
#include <chrono>

#include "utils/gpu_random.h"
#include "fss/gpu_dpf.h"

#include <sytorch/tensor.h>

using T = u64;

int main(int argc, char *argv[])
{
    AESGlobalContext g;
    initAESContext(&g);
    initGPUMemPool();
    int bin = atoi(argv[1]);
    int N = atoi(argv[2]);

    printf("Bin=%d, N=%d\n", bin, N);

    u8 *ptr1, *ptr2;
    getKeyBuf(&ptr1, &ptr2, 50 * OneGB);
    auto keyBuf1 = ptr1;
    auto keyBuf2 = ptr2;
    // auto d_x = (T*) gpuMalloc(N * sizeof(T));
    // checkCudaErrors(hipMemset(d_x, 0, N * sizeof(T)));
    printf("N=%d, memSzN=%lu\n", N, N * sizeof(T));
    initGPURandomness();
    auto d_rin = randomGEOnGpu<T>(N, bin);
    auto h_rin = (T *)moveToCPU((u8 *)d_rin, N * sizeof(T), NULL);
    auto d_X = randomGEOnGpu<T>(N, bin);
    auto h_X = (T *)moveToCPU((u8 *)d_X, N * sizeof(T), NULL);
    // printf("%ld\n", h_X[3]);
    destroyGPURandomness();

    initGPURandomness();
    gpuKeyGenDCF(&keyBuf1, 0, bin, N, d_rin, &g);
    printf("Key size=%lu\n", keyBuf1 - ptr1);
    auto k1 = readGPUDPFKey(&ptr1);

    Stats s;
    auto start = std::chrono::high_resolution_clock::now();
    auto d_O1 = gpuDpf(k1, 0, d_X, &g, &s);
    // gpuDcf<T, 1, idPrologue, idEpilogue>(k1, 0, d_X, &g, (Stats *)&s);
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = end - start;

    printf("Time taken for P0=%lu micros, Transfer time=%lu\n", std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count(), s.transfer_time);

    auto h_O1 = (u32 *)moveToCPU((u8 *)d_O1, k1.memSzOut, (Stats *)NULL);
    gpuFree(d_O1);
    destroyGPURandomness();

    initGPURandomness();
    gpuKeyGenDCF(&keyBuf2, 1, bin, N, d_rin, &g);
    auto k2 = readGPUDPFKey(&ptr2);

    start = std::chrono::high_resolution_clock::now();
    auto d_O2 = gpuDcf<T, 1, idPrologue, idEpilogue>(k2, 1, d_X, &g, (Stats *)NULL);
    end = std::chrono::high_resolution_clock::now();
    elapsed = end - start;

    printf("Time taken for P1=%lu micros\n", std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count());

    auto h_O2 = (u32 *)moveToCPU((u8 *)d_O2, k2.memSzOut, NULL);
    gpuFree(d_O2);
    destroyGPURandomness();

    for (int i = 0; i < N; i++)
    {
        auto o1 = (h_O1[i / 32] >> (i & 31)) & T(1);
        auto o2 = (h_O2[i / 32] >> (i & 31)) & T(1);
        auto o = (o1 + o2) & u32(1);
        if (i < 10 || (o != (h_X[i] < h_rin[i])))
            printf("%d: %u, %u, %u, %lu, %lu\n", i, o1, o2, o, h_X[i], h_rin[i]);
        // assert((h_O1[i] ^ h_O2[i]) == u32(0));
        // assert(o == (h_X[i] < h_rin[i]));
        assert(o == (h_X[i] < h_rin[i]));
    }
    return 0;
}