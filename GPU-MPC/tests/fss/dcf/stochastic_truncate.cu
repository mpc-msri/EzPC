#include <cassert>
#include <cstdint>

#include "utils/gpu_file_utils.h"
#include "utils/misc_utils.h"
#include "utils/gpu_comms.h"
#include "utils/gpu_mem.h"
#include "utils/gpu_random.h"

#include "fss/dcf/gpu_truncate.h"

using T = u64;

int main(int argc, char *argv[]) {
    AESGlobalContext g;
    initAESContext(&g);
    initGPURandomness();
    // initCommBufs(true);
    int bin = 64;
    int bout = 64;
    int shift = 5;
    int N = atoi(argv[3]);
    int party = atoi(argv[1]);
    
    auto peer = new GpuPeer(false);
    peer->connect(party, argv[2]);

    T *h_I;
    auto d_inputMask = randomGEOnGpu<T>(N, bin);
    // checkCudaErrors(hipMemset(d_inputMask, 0, N * sizeof(T)));
    auto h_inputMask = (T*) moveToCPU((u8*) d_inputMask, N * sizeof(T), NULL);
    auto d_masked_I = getMaskedInputOnGpu(N, bin, d_inputMask, &h_I);

    u8 *startPtr, *curPtr;
    size_t keyBufSz = 10 * OneGB;
    getKeyBuf(&startPtr, &curPtr, keyBufSz);
    T* h_r = (T*) cpuMalloc(N * sizeof(T));
    auto d_outputMask = dcf::genGPUStochasticTruncateKey(&curPtr, party, bin, bout, shift, N, d_inputMask, &g, h_r);
    assert(curPtr - startPtr < keyBufSz);
    auto h_outputMask = (T*) moveToCPU((u8*) d_outputMask, N * sizeof(T), NULL);
    gpuFree(d_outputMask);

    curPtr = startPtr;
    auto k = dcf::readGPUTrStochasticKey<T>(&curPtr);

    dcf::gpuStochasticTruncate(k, party, peer, d_masked_I, &g, (Stats*) NULL);

    auto h_O = (T*) moveToCPU((u8*) d_masked_I, N * sizeof(T), NULL);
    checkTrStochastic(bin, bout, shift, N, h_O, h_outputMask, h_I, h_r);
}