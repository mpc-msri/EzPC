#include "hip/hip_runtime.h"
#include "utils/gpu_data_types.h"
#include "utils/gpu_file_utils.h"
#include "utils/misc_utils.h"
#include "utils/gpu_mem.h"
#include "utils/gpu_random.h"
#include "utils/gpu_comms.h"

#include "fss/dcf/gpu_relu.h"

#include <cassert>
#include <sytorch/tensor.h>

using T = u64;

int main(int argc, char *argv[])
{
    // initCommBufs(true);
    initGPUMemPool();
    AESGlobalContext g;
    initAESContext(&g);
    int bw = 64;
    int N = atoi(argv[3]); 
    int party = atoi(argv[1]);

    auto peer = new GpuPeer(false);
    peer->connect(party, argv[2]);

    uint8_t *startPtr, *curPtr;
    getKeyBuf(&startPtr, &curPtr, 10 * OneGB);

    initGPURandomness();
    auto d_mask_X = randomGEOnGpu<T>(N, bw);
    auto h_mask_X = (T *)moveToCPU((u8 *)d_mask_X, N * sizeof(T), NULL);
    T *h_X;
    auto d_masked_X = getMaskedInputOnGpu(N, bw, d_mask_X, &h_X);

    auto d_temp = dcf::gpuGenTwoRoundReluKey(&curPtr, party, bw, bw, N, d_mask_X, &g);
    printf("Key size=%lu\n", curPtr - startPtr);

    auto d_dreluMask = d_temp.first;
    auto d_reluMask = d_temp.second;

    auto h_mask_O = (T *)moveToCPU((u8 *)d_reluMask, N * sizeof(T), NULL);
    auto k1 = dcf::readTwoRoundReluKey<T>(&startPtr);
    T *d_O;
    for (int i = 0; i < 10; i++)
    {
        peer->sync();
        auto start = std::chrono::high_resolution_clock::now();
        auto d_temp = dcf::gpuTwoRoundRelu(peer, party, k1, d_masked_X, &g, (Stats *)NULL);
        auto d_drelu = d_temp.first;
        gpuFree(d_drelu);
        d_O = d_temp.second;
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed = end - start;
        printf("Time taken=%lu micros\n", std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count());
    }

    auto h_O = (T *)moveToCPU((uint8_t *)d_O, N * sizeof(T), (Stats *)NULL);
    gpuFree(d_O);
    destroyGPURandomness();

    printf("bw=%d, bw=%d, N=%d\n", bw, bw, N);
    for (int i = 0; i < N; i++)
    {
        auto unmasked_O = (h_O[i] - h_mask_O[i]);
        cpuMod(unmasked_O, bw);
        auto o = h_X[i] * (1 - (h_X[i] >> (bw - 1)));
        cpuMod(o, bw);
        // auto o = std::min(std::abs((i64) h_X[i]), (i64) max);
        if (i < 10)
            printf("%d: %ld, %ld, %ld, %ld\n", i, h_X[i], o, unmasked_O, h_mask_X[i]);
        assert(o == unmasked_O);
    }

    return 0;
}