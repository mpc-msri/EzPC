// Author: Neha Jawalkar
// Copyright:
// 
// Copyright (c) 2024 Microsoft Research
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <cassert>

#include "utils/gpu_file_utils.h"
#include "utils/misc_utils.h"
#include "utils/gpu_comms.h"
#include "utils/gpu_mem.h"
#include "utils/gpu_random.h"

#include "fss/dcf/gpu_maxpool.h"

#include <llama/array.h>

using T = u32;

template <typename T>
void maxPool2D(MaxpoolParams p, T *in, T *out, T *incomingGrad, T *outgoingGrad)
{
    memset(outgoingGrad, 0, p.N * p.imgH * p.imgW * p.C * sizeof(T));
    for (int i = 0; i < p.N; i++)
    {
        for (int j = 0; j < p.H; j++)
        {
            for (int k = 0; k < p.W; k++)
            {
                for (int l = 0; l < p.C; l++)
                {
                    u64 M = 0;
                    u64 maxIdxI = 0;
                    u64 maxIdxJ = 0;
                    int leftTopCornerH = j * p.strideH - p.zPadHLeft;
                    int leftTopCornerW = k * p.strideW - p.zPadWLeft;
                    for (int m = 0; m < p.FH; m++)
                    {
                        for (int n = 0; n < p.FW; n++)
                        {
                            u64 val = 0;
                            int posH = leftTopCornerH + m;
                            int posW = leftTopCornerW + n;
                            if (posH >= 0 && posH <= p.imgH && posW >= 0 && posW <= p.imgW)
                            {
                                // printf("%d, %d, %d, %d\n", i, posH, posW, l);
                                val = Arr4DIdx(in, p.N, p.imgH, p.imgW, p.C, i, posH, posW, l);
                            }
                            // printf("Val=%lu, %d\n", val, i * p.imgH * p.imgW * p.C + posH * p.imgW * p.C + posW * p.C + l);
                            if (m == 0 && n == 0)
                                M = val;
                            else if (((val - M) & ((T(1) << p.bin) - 1)) < (T(1) << (p.bin - 1)))
                            {
                                M = val;
                                maxIdxI = m;
                                maxIdxJ = n;
                            }
                        }
                    }
                    Arr4DIdx(out, p.N, p.H, p.W, p.C, i, j, k, l) = M;
                    auto inGrad = Arr4DIdx(incomingGrad, p.N, p.H, p.W, p.C, i, j, k, l);
                    auto gradSum = Arr4DIdx(outgoingGrad, p.N, p.imgH, p.imgW, p.C, i, j * p.strideH + maxIdxI, k * p.strideW + maxIdxJ, l);
                    gradSum = (gradSum + inGrad);
                    cpuMod(gradSum, p.bwBackprop);
                    Arr4DIdx(outgoingGrad, p.N, p.imgH, p.imgW, p.C, i, j * p.strideH + maxIdxI, k * p.strideW + maxIdxJ, l) = gradSum;
                    // printf("maxI, maxJ = %d, %d\n", maxIdxI, maxIdxJ);
                }
            }
        }
    }
}

int main(int argc, char *argv[])
{
    AESGlobalContext g;
    initAESContext(&g);
    initGPURandomness();
    // initCommBufs(true);
    int bin = 20;
    int bout = 20;
    int bwBackprop = 32;
    int N = 100;
    int imgH = 30;
    int imgW = 30;
    int C = 3;
    int FH = 5;
    int FW = 5;
    int strideH = 2;
    int strideW = 2;
    int zPadHLeft = 0;
    int zPadHRight = 0;
    int zPadWLeft = 0;
    int zPadWRight = 0;
    bool useMomentum = true;
    int party = atoi(argv[1]);

    auto peer = new GpuPeer(false);
    peer->connect(party, argv[2]);

    MaxpoolParams p = {bin, bout, };
    int inSz = getInSz(p);
    int outSz = getMSz(p);

    T *h_I, *h_incomingGrad;
    auto d_inputMask = randomGEOnGpu<T>(inSz, bin);
    // checkCudaErrors(hipMemset(d_inputMask, 0, inSz * sizeof(T)));
    auto h_inputMask = (T *)moveToCPU((u8 *)d_inputMask, inSz * sizeof(T), NULL);
    auto d_masked_I = getMaskedInputOnGpu(inSz, bin, d_inputMask, &h_I);

    u8 *startPtr, *curPtr;
    getKeyBuf(&startPtr, &curPtr, 4 * OneGB);

    auto d_outputMask = dcf::gpuKeygenMaxpool(&startPtr, party, p, d_inputMask, (u8*) NULL, &g);
    auto h_outputMask = (T *)moveToCPU((u8 *)d_outputMask, outSz * sizeof(T), NULL);

    curPtr = startPtr;
    auto k = dcf::readGPUMaxpoolKey<T>(p, &curPtr);

    auto d_O = dcf::gpuMaxPool(peer, party, p, k, d_masked_I, (u32*) NULL, &g, (Stats*) NULL);
    auto h_O = (T *)moveToCPU((u8 *)d_O, outSz * sizeof(T), NULL);

    T *ct_o = new T[outSz];
    maxPool2D(p, h_I, ct_o, h_incomingGrad, outgoingGradCt);
    for (int i = 0; i < outSz; i++)
    {
        auto unmasked_output = (h_O[i] - h_outputMask[i]);
        cpuMod(unmasked_output, bout);
        if (i < 10 || unmasked_output != ct_o[i])
            printf("%d=%lu %lu\n", i, unmasked_output, ct_o[i]);

        assert(unmasked_output == ct_o[i]);
    }
    return 0;
}